#include "hip/hip_runtime.h"
/* - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - **\ 
 * @authors Aaron Oziel, Sean Blackbourn 
 *
 * Fumitaka Kawasaki (5/3/14):
 * All functions were completed and working. Therefore, the followng comments
 * were removed. 
 *
 * Aaron Wrote (2/3/14):
 * All comments are now tracking progress in conversion from old GpuSim_struct.cu
 * file to the new one here. This is a quick key to keep track of their meanings. 
 *
 *	TODO = 	Needs work and/or is blank. Used to indicate possibly problematic 
 *				functions. 
 *	DONE = 	Likely complete functions. Will still need to be checked for
 *				variable continuity and proper arguments. 
 *   REMOVED =	Deleted, likely due to it becoming unnecessary or not necessary 
 *				for GPU implementation. These functions will likely have to be 
 *				removed from the Model super class.
 *    COPIED = 	These functions were in the original GpuSim_struct.cu file 
 *				and were directly copy-pasted across to this file. 
 *
 \** - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - **/

#include "GPUSpikingModel.h"

#ifdef PERFORMANCE_METRICS
float g_time;
hipEvent_t start, stop;
#endif // PERFORMANCE_METRICS

__constant__ int d_debug_mask[1];

// ----------------------------------------------------------------------------

GPUSpikingModel::GPUSpikingModel(Connections *conns, IAllNeurons *neurons, IAllSynapses *synapses, Layout *layout) : 	
  Model::Model(conns, neurons, synapses, layout),
  synapseIndexMapDevice(NULL),
  randNoise_d(NULL),
  m_allNeuronsDevice(NULL),
  m_allSynapsesDevice(NULL)
{
}

GPUSpikingModel::~GPUSpikingModel() 
{
  //Let Model base class handle de-allocation
}

/*
 * Allocates and initializes memories on CUDA device.
 *
 * @param[out] allNeuronsDevice          Memory loation of the pointer to the neurons list on device memory.
 * @param[out] allSynapsesDevice         Memory loation of the pointer to the synapses list on device memory.
 * @param[in]  sim_info			Pointer to the simulation information.
 */
void GPUSpikingModel::allocDeviceStruct(void** allNeuronsDevice, void** allSynapsesDevice, SimulationInfo *sim_info)
{
  // Allocate Neurons and Synapses strucs on GPU device memory
  m_neurons->allocNeuronDeviceStruct( allNeuronsDevice, sim_info );
  m_synapses->allocSynapseDeviceStruct( allSynapsesDevice, sim_info );

  // Allocate memory for random noise array
  int neuron_count = sim_info->totalNeurons;
  BGSIZE randNoise_d_size = neuron_count * sizeof (float);	// size of random noise array
  HANDLE_ERROR( hipMalloc ( ( void ** ) &randNoise_d, randNoise_d_size ) );

  // Copy host neuron and synapse arrays into GPU device
  m_neurons->copyNeuronHostToDevice( *allNeuronsDevice, sim_info );
  m_synapses->copySynapseHostToDevice( *allSynapsesDevice, sim_info );

  // allocate synapse inverse map in device memory
  allocSynapseImap( neuron_count );
}

/*
 * Copies device memories to host memories and deallocaes them.
 *
 * @param[out] allNeuronsDevice          Memory loation of the pointer to the neurons list on device memory.
 * @param[out] allSynapsesDevice         Memory loation of the pointer to the synapses list on device memory.
 * @param[in]  sim_info                  Pointer to the simulation information.
 */
void GPUSpikingModel::deleteDeviceStruct(void** allNeuronsDevice, void** allSynapsesDevice, SimulationInfo *sim_info)
{
  // copy device synapse and neuron structs to host memory
  m_neurons->copyNeuronDeviceToHost( *allNeuronsDevice, sim_info );

  // Deallocate device memory
  m_neurons->deleteNeuronDeviceStruct( *allNeuronsDevice, sim_info );

  // copy device synapse and neuron structs to host memory
  m_synapses->copySynapseDeviceToHost( *allSynapsesDevice, sim_info );

  // Deallocate device memory
  m_synapses->deleteSynapseDeviceStruct( *allSynapsesDevice );

  deleteSynapseImap();

  HANDLE_ERROR( hipFree( randNoise_d ) );
}

/*
 *  Sets up the Simulation.
 *
 *  @param  sim_info    SimulationInfo class to read information from.
 */
void GPUSpikingModel::setupSim(SimulationInfo *sim_info)
{
  // Set device ID
  HANDLE_ERROR( hipSetDevice( g_deviceId ) );

  // Set DEBUG flag
  HANDLE_ERROR( hipMemcpyToSymbol(HIP_SYMBOL(d_debug_mask), &g_debug_mask, sizeof(int) ) );

  Model::setupSim(sim_info);

  //initialize Mersenne Twister
  //assuming neuron_count >= 100 and is a multiple of 100. Note rng_mt_rng_count must be <= MT_RNG_COUNT
  int rng_blocks = 25; //# of blocks the kernel will use
  int rng_nPerRng = 4; //# of iterations per thread (thread granularity, # of rands generated per thread)
  int rng_mt_rng_count = sim_info->totalNeurons/rng_nPerRng; //# of threads to generate for neuron_count rand #s
  int rng_threads = rng_mt_rng_count/rng_blocks; //# threads per block needed
  initMTGPU(sim_info->seed, rng_blocks, rng_threads, rng_nPerRng, rng_mt_rng_count);

#ifdef PERFORMANCE_METRICS
  hipEventCreate( &start );
  hipEventCreate( &stop );

  t_gpu_rndGeneration = 0.0;
  t_gpu_advanceNeurons = 0.0;
  t_gpu_advanceSynapses = 0.0;
  t_gpu_calcSummation = 0.0;
#endif // PERFORMANCE_METRICS

  // allocates memories on CUDA device
  allocDeviceStruct((void **)&m_allNeuronsDevice, (void **)&m_allSynapsesDevice, sim_info);

  // copy inverse map to the device memory
  copySynapseIndexMapHostToDevice(*m_synapseIndexMap, sim_info->totalNeurons);

  // set some parameters used for advanceNeuronsDevice
  m_neurons->setAdvanceNeuronsDeviceParams(*m_synapses);

  // set some parameters used for advanceSynapsesDevice
  m_synapses->setAdvanceSynapsesDeviceParams();
}

/* 
 *  Begin terminating the simulator.
 *
 *  @param  sim_info    SimulationInfo to refer.
 */
void GPUSpikingModel::cleanupSim(SimulationInfo *sim_info)
{
  // deallocates memories on CUDA device
  deleteDeviceStruct((void**)&m_allNeuronsDevice, (void**)&m_allSynapsesDevice, sim_info);

#ifdef PERFORMANCE_METRICS
  hipEventDestroy( start );
  hipEventDestroy( stop );
#endif // PERFORMANCE_METRICS
}

/*
 *  Loads the simulation based on istream input.
 *
 *  @param  input   istream to read from.
 *  @param  sim_info    used as a reference to set info for neurons and synapses.
 */
void GPUSpikingModel::deserialize(istream& input, const SimulationInfo *sim_info)
{
  Model::deserialize(input, sim_info);

  // copy inverse map to the device memory
  copySynapseIndexMapHostToDevice(*m_synapseIndexMap, sim_info->totalNeurons);

  // Reinitialize device struct - Copy host neuron and synapse arrays into GPU device
  m_neurons->copyNeuronHostToDevice( m_allNeuronsDevice, sim_info );
  m_synapses->copySynapseHostToDevice( m_allSynapsesDevice, sim_info );
}

/* 
 *  Advance everything in the model one time step. In this case, that
 *  means calling all of the kernels that do the "micro step" updating
 *  (i.e., NOT the stuff associated with growth).
 *
 *  @param  sim_info    SimulationInfo class to read information from.
 */
void GPUSpikingModel::advance(const SimulationInfo *sim_info)
{
#ifdef PERFORMANCE_METRICS
  // Reset CUDA timer to start measurement of GPU operations
  cudaStartTimer();
#endif // PERFORMANCE_METRICS

  normalMTGPU(randNoise_d);

#ifdef PERFORMANCE_METRICS
  cudaLapTime(t_gpu_rndGeneration);
  cudaStartTimer();
#endif // PERFORMANCE_METRICS

  // display running info to console
  // Advance neurons ------------->
  m_neurons->advanceNeurons(*m_synapses, m_allNeuronsDevice, m_allSynapsesDevice, sim_info, randNoise_d, synapseIndexMapDevice);

#ifdef PERFORMANCE_METRICS
  cudaLapTime(t_gpu_advanceNeurons);
  cudaStartTimer();
#endif // PERFORMANCE_METRICS

  // Advance synapses ------------->
  m_synapses->advanceSynapses(m_allSynapsesDevice, m_allNeuronsDevice, synapseIndexMapDevice, sim_info);

#ifdef PERFORMANCE_METRICS
  cudaLapTime(t_gpu_advanceSynapses);
  cudaStartTimer();
#endif // PERFORMANCE_METRICS

  // calculate summation point
  calcSummationMap(sim_info);

#ifdef PERFORMANCE_METRICS
 cudaLapTime(t_gpu_calcSummation);
#endif // PERFORMANCE_METRICS
}

/*
 * Add psr of all incoming synapses to summation points.
 *
 * @param[in] sim_info                   Pointer to the simulation information.
 */
void GPUSpikingModel::calcSummationMap(const SimulationInfo *sim_info)
{
  // CUDA parameters
  const int threadsPerBlock = 256;
  int blocksPerGrid = ( sim_info->totalNeurons + threadsPerBlock - 1 ) / threadsPerBlock;

  calcSummationMapDevice <<< blocksPerGrid, threadsPerBlock >>> ( sim_info->totalNeurons, m_allNeuronsDevice, synapseIndexMapDevice, m_allSynapsesDevice );
}

/* 
 *  Update the connection of all the Neurons and Synapses of the simulation.
 *
 *  @param  sim_info    SimulationInfo class to read information from.
 */
void GPUSpikingModel::updateConnections(const SimulationInfo *sim_info)
{
  dynamic_cast<AllSpikingNeurons*>(m_neurons)->copyNeuronDeviceSpikeCountsToHost(m_allNeuronsDevice, sim_info);
  dynamic_cast<AllSpikingNeurons*>(m_neurons)->copyNeuronDeviceSpikeHistoryToHost(m_allNeuronsDevice, sim_info);

  // Update Connections data
  if (m_conns->updateConnections(*m_neurons, sim_info, m_layout)) {
    m_conns->updateSynapsesWeights(sim_info->totalNeurons, *m_neurons, *m_synapses, sim_info, m_allNeuronsDevice, m_allSynapsesDevice, m_layout);
    // create synapse inverse map
    m_synapses->createSynapseImap(m_synapseIndexMap, sim_info);
    // copy inverse map to the device memory
    copySynapseIndexMapHostToDevice(*m_synapseIndexMap, sim_info->totalNeurons);
  }
}

/*
 *  Update the Neuron's history.
 *
 *  @param  sim_info    SimulationInfo to refer from.
 */
void GPUSpikingModel::updateHistory(const SimulationInfo *sim_info)
{
  Model::updateHistory(sim_info);

  // clear spike count
  dynamic_cast<AllSpikingNeurons*>(m_neurons)->clearNeuronSpikeCounts(m_allNeuronsDevice, sim_info);
}

/* ------------------*\
   |* # Helper Functions
   \* ------------------*/

/*
 *  Allocate device memory for synapse inverse map.
 *  @param  count	The number of neurons.
 */
void GPUSpikingModel::allocSynapseImap( int count )
{
  SynapseIndexMap synapseIndexMap;

  HANDLE_ERROR( hipMalloc( ( void ** ) &synapseIndexMap.outgoingSynapseBegin, count * sizeof( BGSIZE ) ) );
  HANDLE_ERROR( hipMalloc( ( void ** ) &synapseIndexMap.outgoingSynapseCount, count * sizeof( BGSIZE ) ) );
  HANDLE_ERROR( hipMemset(synapseIndexMap.outgoingSynapseBegin, 0, count * sizeof( BGSIZE ) ) );
  HANDLE_ERROR( hipMemset(synapseIndexMap.outgoingSynapseCount, 0, count * sizeof( BGSIZE ) ) );

  HANDLE_ERROR( hipMalloc( ( void ** ) &synapseIndexMap.incomingSynapseBegin, count * sizeof( BGSIZE ) ) );
  HANDLE_ERROR( hipMalloc( ( void ** ) &synapseIndexMap.incomingSynapseCount, count * sizeof( BGSIZE ) ) );
  HANDLE_ERROR( hipMemset(synapseIndexMap.incomingSynapseBegin, 0, count * sizeof( BGSIZE ) ) );
  HANDLE_ERROR( hipMemset(synapseIndexMap.incomingSynapseCount, 0, count * sizeof( BGSIZE ) ) );

  HANDLE_ERROR( hipMalloc( ( void ** ) &synapseIndexMapDevice, sizeof( SynapseIndexMap ) ) );
  HANDLE_ERROR( hipMemcpy( synapseIndexMapDevice, &synapseIndexMap, sizeof( SynapseIndexMap ), hipMemcpyHostToDevice ) );
}

/*
 *  Deallocate device memory for synapse inverse map.
 */
void GPUSpikingModel::deleteSynapseImap(  )
{
  SynapseIndexMap synapseIndexMap;

  HANDLE_ERROR( hipMemcpy ( &synapseIndexMap, synapseIndexMapDevice, sizeof( SynapseIndexMap ), hipMemcpyDeviceToHost ) );

  HANDLE_ERROR( hipFree( synapseIndexMap.outgoingSynapseBegin ) );
  HANDLE_ERROR( hipFree( synapseIndexMap.outgoingSynapseCount ) );
  HANDLE_ERROR( hipFree( synapseIndexMap.outgoingSynapseIndexMap ) );

  HANDLE_ERROR( hipFree( synapseIndexMap.incomingSynapseBegin ) );
  HANDLE_ERROR( hipFree( synapseIndexMap.incomingSynapseCount ) );
  HANDLE_ERROR( hipFree( synapseIndexMap.incomingSynapseIndexMap ) );

  HANDLE_ERROR( hipFree( synapseIndexMapDevice ) );
}

/* 
 *  Copy SynapseIndexMap in host memory to SynapseIndexMap in device memory.
 *
 *  @param  synapseIndexMapHost		Reference to the SynapseIndexMap in host memory.
 *  @param  neuron_count		The number of neurons.
 */
void GPUSpikingModel::copySynapseIndexMapHostToDevice(SynapseIndexMap &synapseIndexMapHost, int neuron_count)
{
  int total_synapse_counts = dynamic_cast<AllSynapses*>(m_synapses)->total_synapse_counts;

  if (total_synapse_counts == 0)
    return;

  SynapseIndexMap synapseIndexMap;

  HANDLE_ERROR( hipMemcpy ( &synapseIndexMap, synapseIndexMapDevice, sizeof( SynapseIndexMap ), hipMemcpyDeviceToHost ) );

  // forward map
  HANDLE_ERROR( hipMemcpy ( synapseIndexMap.outgoingSynapseBegin, synapseIndexMapHost.outgoingSynapseBegin, neuron_count * sizeof( BGSIZE ), hipMemcpyHostToDevice ) );
  HANDLE_ERROR( hipMemcpy ( synapseIndexMap.outgoingSynapseCount, synapseIndexMapHost.outgoingSynapseCount, neuron_count * sizeof( BGSIZE ), hipMemcpyHostToDevice ) );
  // the number of synapses may change, so we reallocate the memory
  if (synapseIndexMap.outgoingSynapseIndexMap != NULL) {
    HANDLE_ERROR( hipFree( synapseIndexMap.outgoingSynapseIndexMap ) );
  }
  HANDLE_ERROR( hipMalloc( ( void ** ) &synapseIndexMap.outgoingSynapseIndexMap, total_synapse_counts * sizeof( BGSIZE ) ) );
  HANDLE_ERROR( hipMemcpy ( synapseIndexMap.outgoingSynapseIndexMap, synapseIndexMapHost.outgoingSynapseIndexMap, total_synapse_counts * sizeof( BGSIZE ), hipMemcpyHostToDevice ) );

  // active synapse map
  HANDLE_ERROR( hipMemcpy ( synapseIndexMap.incomingSynapseBegin, synapseIndexMapHost.incomingSynapseBegin, neuron_count * sizeof( BGSIZE ), hipMemcpyHostToDevice ) );
  HANDLE_ERROR( hipMemcpy ( synapseIndexMap.incomingSynapseCount, synapseIndexMapHost.incomingSynapseCount, neuron_count * sizeof( BGSIZE ), hipMemcpyHostToDevice ) );
  // the number of synapses may change, so we reallocate the memory
  if (synapseIndexMap.incomingSynapseIndexMap != NULL) {
    HANDLE_ERROR( hipFree( synapseIndexMap.incomingSynapseIndexMap ) );
  }
  HANDLE_ERROR( hipMalloc( ( void ** ) &synapseIndexMap.incomingSynapseIndexMap, total_synapse_counts * sizeof( BGSIZE ) ) );
  HANDLE_ERROR( hipMemcpy ( synapseIndexMap.incomingSynapseIndexMap, synapseIndexMapHost.incomingSynapseIndexMap, total_synapse_counts * sizeof( BGSIZE ), hipMemcpyHostToDevice ) );

  HANDLE_ERROR( hipMemcpy ( synapseIndexMapDevice, &synapseIndexMap, sizeof( SynapseIndexMap ), hipMemcpyHostToDevice ) );
}

/* ------------------*\
   |* # Global Functions
   \* ------------------*/

/**
 * Calculate the sum of synaptic input to each neuron.
 *
 * Calculate the sum of synaptic input to each neuron. One thread
 * corresponds to one neuron. Iterates sequentially through the
 * forward synapse index map (synapseIndexMapDevice) to access only
 * existing synapses. Using this structure eliminates the need to skip
 * synapses that have undergone lazy deletion from the main
 * (allSynapsesDevice) synapse structure. The forward map is
 * re-computed during each network restructure (once per epoch) to
 * ensure that all synapse pointers for a neuron are stored
 * contiguously.
 * 
 * @param[in] totalNeurons           Number of neurons in the entire simulation.
 * @param[in,out] allNeuronsDevice   Pointer to Neuron structures in device memory.
 * @param[in] synapseIndexMapDevice  Pointer to forward map structures in device memory.
 * @param[in] allSynapsesDevice      Pointer to Synapse structures in device memory.
 */
__global__ void calcSummationMapDevice(int totalNeurons, 
				       AllSpikingNeuronsDeviceProperties* __restrict__ allNeuronsDevice, 
				       const SynapseIndexMap* __restrict__ synapseIndexMapDevice, 
				       const AllSpikingSynapsesDeviceProperties* __restrict__ allSynapsesDevice)
{
  // The usual thread ID calculation and guard against excess threads
  // (beyond the number of neurons, in this case).
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if ( idx >= totalNeurons )
    return;

  // Number of incoming synapses
  const BGSIZE synCount = synapseIndexMapDevice->incomingSynapseCount[idx];
  // Optimization: terminate thread if no incoming synapses
  if (synCount != 0) {
    // Index of start of this neuron's block of forward map entries
    const int beginIndex = synapseIndexMapDevice->incomingSynapseBegin[idx];
    // Address of the start of this neuron's block of forward map entries
    const BGSIZE* activeMap_begin = 
      &(synapseIndexMapDevice->incomingSynapseIndexMap[beginIndex]);
    // Summed postsynaptic response (PSR)
    BGFLOAT sum = 0.0;
    // Index of the current incoming synapse
    BGSIZE synIndex;
    // Repeat for each incoming synapse
    for (BGSIZE i = 0; i < synCount; i++) {
      // Get index of current incoming synapse
      synIndex = activeMap_begin[i];
      // Fetch its PSR and add into sum
      sum += allSynapsesDevice->psr[synIndex];
    }
    // Store summed PSR into this neuron's summation point
    allNeuronsDevice->summation_map[idx] = sum;
  }
}

