/*
 * AllIFNeurons_d.cu
 *
 */

#include "AllIFNeurons.h"
#include "Book.h"

/*
 *  Allocate GPU memories to store all neurons' states,
 *  and copy them from host to GPU memory.
 *
 *  @param  allNeuronsDevice   Reference to the AllIFNeuronsDeviceProperties struct on device memory.
 *  @param  sim_info           SimulationInfo to refer from.
 */
void AllIFNeurons::allocNeuronDeviceStruct( void** allNeuronsDevice, SimulationInfo *sim_info ) {
	AllIFNeuronsDeviceProperties allNeurons;

	allocDeviceStruct( allNeurons, sim_info );

        HANDLE_ERROR( hipMalloc( allNeuronsDevice, sizeof( AllIFNeuronsDeviceProperties ) ) );
        HANDLE_ERROR( hipMemcpy ( *allNeuronsDevice, &allNeurons, sizeof( AllIFNeuronsDeviceProperties ), hipMemcpyHostToDevice ) );
}

/*
 *  Allocate GPU memories to store all neurons' states.
 *  (Helper function of allocNeuronDeviceStruct)
 *
 *  @param  allNeurons         Reference to the AllIFNeuronsDeviceProperties struct.
 *  @param  sim_info           SimulationInfo to refer from.
 */
void AllIFNeurons::allocDeviceStruct( AllIFNeuronsDeviceProperties &allNeurons, SimulationInfo *sim_info ) {
	int count = sim_info->totalNeurons;
	int max_spikes = static_cast<int> (sim_info->epochDuration * sim_info->maxFiringRate);
 
	HANDLE_ERROR( hipMalloc( ( void ** ) &allNeurons.C1, count * sizeof( BGFLOAT ) ) );
	HANDLE_ERROR( hipMalloc( ( void ** ) &allNeurons.C2, count * sizeof( BGFLOAT ) ) );
	HANDLE_ERROR( hipMalloc( ( void ** ) &allNeurons.Cm, count * sizeof( BGFLOAT ) ) );
	HANDLE_ERROR( hipMalloc( ( void ** ) &allNeurons.I0, count * sizeof( BGFLOAT ) ) );
	HANDLE_ERROR( hipMalloc( ( void ** ) &allNeurons.Iinject, count * sizeof( BGFLOAT ) ) );
	HANDLE_ERROR( hipMalloc( ( void ** ) &allNeurons.Inoise, count * sizeof( BGFLOAT ) ) );
	HANDLE_ERROR( hipMalloc( ( void ** ) &allNeurons.Isyn, count * sizeof( BGFLOAT ) ) );
	HANDLE_ERROR( hipMalloc( ( void ** ) &allNeurons.Rm, count * sizeof( BGFLOAT ) ) );
	HANDLE_ERROR( hipMalloc( ( void ** ) &allNeurons.Tau, count * sizeof( BGFLOAT ) ) );
	HANDLE_ERROR( hipMalloc( ( void ** ) &allNeurons.Trefract, count * sizeof( BGFLOAT ) ) );
	HANDLE_ERROR( hipMalloc( ( void ** ) &allNeurons.Vinit, count * sizeof( BGFLOAT ) ) );
	HANDLE_ERROR( hipMalloc( ( void ** ) &allNeurons.Vm, count * sizeof( BGFLOAT ) ) );
	HANDLE_ERROR( hipMalloc( ( void ** ) &allNeurons.Vreset, count * sizeof( BGFLOAT ) ) );
	HANDLE_ERROR( hipMalloc( ( void ** ) &allNeurons.Vrest, count * sizeof( BGFLOAT ) ) );
	HANDLE_ERROR( hipMalloc( ( void ** ) &allNeurons.Vthresh, count * sizeof( BGFLOAT ) ) );
	HANDLE_ERROR( hipMalloc( ( void ** ) &allNeurons.hasFired, count * sizeof( bool ) ) );
	HANDLE_ERROR( hipMalloc( ( void ** ) &allNeurons.nStepsInRefr, count * sizeof( int ) ) );
	HANDLE_ERROR( hipMalloc( ( void ** ) &allNeurons.spikeCount, count * sizeof( int ) ) );
	HANDLE_ERROR( hipMalloc( ( void ** ) &allNeurons.spikeCountOffset, count * sizeof( int ) ) );
	HANDLE_ERROR( hipMalloc( ( void ** ) &allNeurons.summation_map, count * sizeof( BGFLOAT ) ) );
	HANDLE_ERROR( hipMalloc( ( void ** ) &allNeurons.spike_history, count * sizeof( uint64_t* ) ) );
	
	uint64_t* pSpikeHistory[count];
	for (int i = 0; i < count; i++) {
		HANDLE_ERROR( hipMalloc( ( void ** ) &pSpikeHistory[i], max_spikes * sizeof( uint64_t ) ) );
	}
	HANDLE_ERROR( hipMemcpy ( allNeurons.spike_history, pSpikeHistory,
		count * sizeof( uint64_t* ), hipMemcpyHostToDevice ) );

	// get device summation point address and set it to sim info
	sim_info->pSummationMap = allNeurons.summation_map;
}

/*
 *  Delete GPU memories.
 *
 *  @param  allNeuronsDevice   Reference to the AllIFNeuronsDeviceProperties struct on device memory.
 *  @param  sim_info           SimulationInfo to refer from.
 */
void AllIFNeurons::deleteNeuronDeviceStruct( void* allNeuronsDevice, const SimulationInfo *sim_info ) {
	AllIFNeuronsDeviceProperties allNeurons;

	HANDLE_ERROR( hipMemcpy ( &allNeurons, allNeuronsDevice, sizeof( AllIFNeuronsDeviceProperties ), hipMemcpyDeviceToHost ) );

	deleteDeviceStruct( allNeurons, sim_info );

	HANDLE_ERROR( hipFree( allNeuronsDevice ) );
}

/*
 *  Delete GPU memories.
 *  (Helper function of deleteNeuronDeviceStruct)
 *
 *  @param  allNeurons         Reference to the AllIFNeuronsDeviceProperties struct.
 *  @param  sim_info           SimulationInfo to refer from.
 */
void AllIFNeurons::deleteDeviceStruct( AllIFNeuronsDeviceProperties& allNeurons, const SimulationInfo *sim_info ) {
	int count = sim_info->totalNeurons;

	uint64_t* pSpikeHistory[count];
	HANDLE_ERROR( hipMemcpy ( pSpikeHistory, allNeurons.spike_history,
		count * sizeof( uint64_t* ), hipMemcpyDeviceToHost ) );
	for (int i = 0; i < count; i++) {
		HANDLE_ERROR( hipFree( pSpikeHistory[i] ) );
	}

	HANDLE_ERROR( hipFree( allNeurons.C1 ) );
	HANDLE_ERROR( hipFree( allNeurons.C2 ) );
	HANDLE_ERROR( hipFree( allNeurons.Cm ) );
	HANDLE_ERROR( hipFree( allNeurons.I0 ) );
	HANDLE_ERROR( hipFree( allNeurons.Iinject ) );
	HANDLE_ERROR( hipFree( allNeurons.Inoise ) );
	HANDLE_ERROR( hipFree( allNeurons.Isyn ) );
	HANDLE_ERROR( hipFree( allNeurons.Rm ) );
	HANDLE_ERROR( hipFree( allNeurons.Tau ) );
	HANDLE_ERROR( hipFree( allNeurons.Trefract ) );
	HANDLE_ERROR( hipFree( allNeurons.Vinit ) );
	HANDLE_ERROR( hipFree( allNeurons.Vm ) );
	HANDLE_ERROR( hipFree( allNeurons.Vreset ) );
	HANDLE_ERROR( hipFree( allNeurons.Vrest ) );
	HANDLE_ERROR( hipFree( allNeurons.Vthresh ) );
	HANDLE_ERROR( hipFree( allNeurons.hasFired ) );
	HANDLE_ERROR( hipFree( allNeurons.nStepsInRefr ) );
	HANDLE_ERROR( hipFree( allNeurons.spikeCount ) );
	HANDLE_ERROR( hipFree( allNeurons.spikeCountOffset ) );
	HANDLE_ERROR( hipFree( allNeurons.summation_map ) );
	HANDLE_ERROR( hipFree( allNeurons.spike_history ) );
}

/*
 *  Copy all neurons' data from host to device.
 *
 *  @param  allNeuronsDevice   Reference to the AllIFNeuronsDeviceProperties struct on device memory.
 *  @param  sim_info           SimulationInfo to refer from.
 */
void AllIFNeurons::copyNeuronHostToDevice( void* allNeuronsDevice, const SimulationInfo *sim_info ) { 
	AllIFNeuronsDeviceProperties allNeurons;

	HANDLE_ERROR( hipMemcpy ( &allNeurons, allNeuronsDevice, sizeof( AllIFNeuronsDeviceProperties ), hipMemcpyDeviceToHost ) );
	copyHostToDevice( allNeurons, sim_info );
}

/*
 *  Copy all neurons' data from host to device.
 *  (Helper function of copyNeuronHostToDevice)
 *
 *  @param  allNeurons         Reference to the AllIFNeuronsDeviceProperties struct.
 *  @param  sim_info           SimulationInfo to refer from.
 */
void AllIFNeurons::copyHostToDevice( AllIFNeuronsDeviceProperties& allNeurons, const SimulationInfo *sim_info ) { 
	int count = sim_info->totalNeurons;

	HANDLE_ERROR( hipMemcpy ( allNeurons.C1, C1, count * sizeof( BGFLOAT ), hipMemcpyHostToDevice ) );
	HANDLE_ERROR( hipMemcpy ( allNeurons.C2, C2, count * sizeof( BGFLOAT ), hipMemcpyHostToDevice ) );
	HANDLE_ERROR( hipMemcpy ( allNeurons.Cm, Cm, count * sizeof( BGFLOAT ), hipMemcpyHostToDevice ) );
	HANDLE_ERROR( hipMemcpy ( allNeurons.I0, I0, count * sizeof( BGFLOAT ), hipMemcpyHostToDevice ) );
	HANDLE_ERROR( hipMemcpy ( allNeurons.Iinject, Iinject, count * sizeof( BGFLOAT ), hipMemcpyHostToDevice ) );
	HANDLE_ERROR( hipMemcpy ( allNeurons.Inoise, Inoise, count * sizeof( BGFLOAT ), hipMemcpyHostToDevice ) );
	HANDLE_ERROR( hipMemcpy ( allNeurons.Isyn, Isyn, count * sizeof( BGFLOAT ), hipMemcpyHostToDevice ) );
	HANDLE_ERROR( hipMemcpy ( allNeurons.Rm, Rm, count * sizeof( BGFLOAT ), hipMemcpyHostToDevice ) );
	HANDLE_ERROR( hipMemcpy ( allNeurons.Tau, Tau, count * sizeof( BGFLOAT ), hipMemcpyHostToDevice ) );
	HANDLE_ERROR( hipMemcpy ( allNeurons.Trefract, Trefract, count * sizeof( BGFLOAT ), hipMemcpyHostToDevice ) );
	HANDLE_ERROR( hipMemcpy ( allNeurons.Vinit, Vinit, count * sizeof( BGFLOAT ), hipMemcpyHostToDevice ) );
	HANDLE_ERROR( hipMemcpy ( allNeurons.Vm, Vm, count * sizeof( BGFLOAT ), hipMemcpyHostToDevice ) );
	HANDLE_ERROR( hipMemcpy ( allNeurons.Vreset, Vreset, count * sizeof( BGFLOAT ), hipMemcpyHostToDevice ) );
	HANDLE_ERROR( hipMemcpy ( allNeurons.Vrest, Vrest, count * sizeof( BGFLOAT ), hipMemcpyHostToDevice ) );
	HANDLE_ERROR( hipMemcpy ( allNeurons.Vthresh, Vthresh, count * sizeof( BGFLOAT ), hipMemcpyHostToDevice ) );
	HANDLE_ERROR( hipMemcpy ( allNeurons.hasFired, hasFired, count * sizeof( bool ), hipMemcpyHostToDevice ) );
	HANDLE_ERROR( hipMemcpy ( allNeurons.nStepsInRefr, nStepsInRefr, count * sizeof( int ), hipMemcpyHostToDevice ) );
	HANDLE_ERROR( hipMemcpy ( allNeurons.spikeCount, spikeCount, count * sizeof( int ), hipMemcpyHostToDevice ) );
	HANDLE_ERROR( hipMemcpy ( allNeurons.spikeCountOffset, spikeCountOffset, count * sizeof( int ), hipMemcpyHostToDevice ) );

        int max_spikes = static_cast<int> (sim_info->epochDuration * sim_info->maxFiringRate);
        uint64_t* pSpikeHistory[count];
        HANDLE_ERROR( hipMemcpy ( pSpikeHistory, allNeurons.spike_history, count * sizeof( uint64_t* ), hipMemcpyDeviceToHost ) );
        for (int i = 0; i < count; i++) {
                HANDLE_ERROR( hipMemcpy ( pSpikeHistory[i], spike_history[i], max_spikes * sizeof( uint64_t ), hipMemcpyHostToDevice ) );
        }
}

/*
 *  Copy all neurons' data from device to host.
 *
 *  @param  allNeuronsDevice   Reference to the AllIFNeuronsDeviceProperties struct on device memory.
 *  @param  sim_info           SimulationInfo to refer from.
 */
void AllIFNeurons::copyNeuronDeviceToHost( void* allNeuronsDevice, const SimulationInfo *sim_info ) {
	AllIFNeuronsDeviceProperties allNeurons;

	HANDLE_ERROR( hipMemcpy ( &allNeurons, allNeuronsDevice, sizeof( AllIFNeuronsDeviceProperties ), hipMemcpyDeviceToHost ) );
	copyDeviceToHost( allNeurons, sim_info );
}

/*
 *  Copy all neurons' data from device to host.
 *  (Helper function of copyNeuronDeviceToHost)
 *
 *  @param  allNeurons         Reference to the AllIFNeuronsDeviceProperties struct.
 *  @param  sim_info           SimulationInfo to refer from.
 */
void AllIFNeurons::copyDeviceToHost( AllIFNeuronsDeviceProperties& allNeurons, const SimulationInfo *sim_info ) {
	int count = sim_info->totalNeurons;

	HANDLE_ERROR( hipMemcpy ( C1, allNeurons.C1, count * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
	HANDLE_ERROR( hipMemcpy ( C2, allNeurons.C2, count * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
	HANDLE_ERROR( hipMemcpy ( Cm, allNeurons.C1, count * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
	HANDLE_ERROR( hipMemcpy ( I0, allNeurons.I0, count * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
	HANDLE_ERROR( hipMemcpy ( Iinject, allNeurons.Iinject, count * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
	HANDLE_ERROR( hipMemcpy ( Inoise, allNeurons.Inoise, count * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
	HANDLE_ERROR( hipMemcpy ( Isyn, allNeurons.Isyn, count * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
	HANDLE_ERROR( hipMemcpy ( Rm, allNeurons.Rm, count * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
	HANDLE_ERROR( hipMemcpy ( Tau, allNeurons.Tau, count * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
	HANDLE_ERROR( hipMemcpy ( Trefract, allNeurons.Trefract, count * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
	HANDLE_ERROR( hipMemcpy ( Vinit, allNeurons.Vinit, count * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
	HANDLE_ERROR( hipMemcpy ( Vm, allNeurons.Vm, count * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
	HANDLE_ERROR( hipMemcpy ( Vreset, allNeurons.Vreset, count * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
	HANDLE_ERROR( hipMemcpy ( Vrest, allNeurons.Vrest, count * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
	HANDLE_ERROR( hipMemcpy ( Vthresh, allNeurons.Vthresh, count * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
	HANDLE_ERROR( hipMemcpy ( hasFired, allNeurons.hasFired, count * sizeof( bool ), hipMemcpyDeviceToHost ) );
	HANDLE_ERROR( hipMemcpy ( nStepsInRefr, allNeurons.nStepsInRefr, count * sizeof( int ), hipMemcpyDeviceToHost ) );
	HANDLE_ERROR( hipMemcpy ( spikeCount, allNeurons.spikeCount, count * sizeof( int ), hipMemcpyDeviceToHost ) );
	HANDLE_ERROR( hipMemcpy ( spikeCountOffset, allNeurons.spikeCountOffset, count * sizeof( int ), hipMemcpyDeviceToHost ) );

        int max_spikes = static_cast<int> (sim_info->epochDuration * sim_info->maxFiringRate);
        uint64_t* pSpikeHistory[count];
        HANDLE_ERROR( hipMemcpy ( pSpikeHistory, allNeurons.spike_history, count * sizeof( uint64_t* ), hipMemcpyDeviceToHost ) );
        for (int i = 0; i < count; i++) {
                HANDLE_ERROR( hipMemcpy ( spike_history[i], pSpikeHistory[i], max_spikes * sizeof( uint64_t ), hipMemcpyDeviceToHost ) );
        }
}

/*
 *  Copy spike history data stored in device memory to host.
 *
 *  @param  allNeuronsDevice   Reference to the AllIFNeuronsDeviceProperties struct on device memory.
 *  @param  sim_info           SimulationInfo to refer from.
 */
void AllIFNeurons::copyNeuronDeviceSpikeHistoryToHost( void* allNeuronsDevice, const SimulationInfo *sim_info ) 
{        
        AllIFNeuronsDeviceProperties allNeurons;
        HANDLE_ERROR( hipMemcpy ( &allNeurons, allNeuronsDevice, sizeof( AllIFNeuronsDeviceProperties ), hipMemcpyDeviceToHost ) );        
        AllSpikingNeurons::copyDeviceSpikeHistoryToHost( allNeurons, sim_info );
}

/*
 *  Copy spike counts data stored in device memory to host.
 *
 *  @param  allNeuronsDevice   Reference to the AllIFNeuronsDeviceProperties struct on device memory.
 *  @param  sim_info           SimulationInfo to refer from.
 */
void AllIFNeurons::copyNeuronDeviceSpikeCountsToHost( void* allNeuronsDevice, const SimulationInfo *sim_info )
{
        AllIFNeuronsDeviceProperties allNeurons;
        HANDLE_ERROR( hipMemcpy ( &allNeurons, allNeuronsDevice, sizeof( AllIFNeuronsDeviceProperties ), hipMemcpyDeviceToHost ) );
        AllSpikingNeurons::copyDeviceSpikeCountsToHost( allNeurons, sim_info );
}

/*
 *  Clear the spike counts out of all neurons.
 *
 *  @param  allNeuronsDevice   Reference to the AllIFNeuronsDeviceProperties struct on device memory.
 *  @param  sim_info           SimulationInfo to refer from.
 */
void AllIFNeurons::clearNeuronSpikeCounts( void* allNeuronsDevice, const SimulationInfo *sim_info )
{
        AllIFNeuronsDeviceProperties allNeurons;
        HANDLE_ERROR( hipMemcpy ( &allNeurons, allNeuronsDevice, sizeof( AllIFNeuronsDeviceProperties ), hipMemcpyDeviceToHost ) );
        AllSpikingNeurons::clearDeviceSpikeCounts( allNeurons, sim_info );
}

/*
 *  Update the state of all neurons for a time step
 *  Notify outgoing synapses if neuron has fired.
 *
 *  @param  synapses               Reference to the allSynapses struct on host memory.
 *  @param  allNeuronsDevice       Reference to the AllIFNeuronsDeviceProperties struct 
 *                                 on device memory.
 *  @param  allSynapsesDevice      Reference to the allSynapsesDeviceProperties struct 
 *                                 on device memory.
 *  @param  sim_info               SimulationInfo to refer from.
 *  @param  randNoise              Reference to the random noise array.
 *  @param  synapseIndexMapDevice  Reference to the SynapseIndexMap on device memory.
 */
void AllIFNeurons::advanceNeurons( IAllSynapses &synapses, void* allNeuronsDevice, void* allSynapsesDevice, const SimulationInfo *sim_info, float* randNoise, SynapseIndexMap* synapseIndexMapDevice )
{
}
