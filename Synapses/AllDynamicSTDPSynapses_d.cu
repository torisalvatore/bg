/*
 * AllDynamicSTDPSynapses_d.cu
 *
 */

#include "AllDynamicSTDPSynapses.h"
#include "AllSynapsesDeviceFuncs.h"
#include "Book.h"

/*
 *  Allocate GPU memories to store all synapses' states,
 *  and copy them from host to GPU memory.
 *
 *  @param  allSynapsesDevice  Reference to the AllDynamicSTDPSynapsesDeviceProperties struct 
 *                             on device memory.
 *  @param  sim_info           SimulationInfo to refer from.
 */
void AllDynamicSTDPSynapses::allocSynapseDeviceStruct( void** allSynapsesDevice, const SimulationInfo *sim_info ) {
	allocSynapseDeviceStruct( allSynapsesDevice, sim_info->totalNeurons, sim_info->maxSynapsesPerNeuron );
}

/*
 *  Allocate GPU memories to store all synapses' states,
 *  and copy them from host to GPU memory.
 *
 *  @param  allSynapsesDevice     Reference to the AllDynamicSTDPSynapsesDeviceProperties struct 
 *                                on device memory.
 *  @param  num_neurons           Number of neurons.
 *  @param  maxSynapsesPerNeuron  Maximum number of synapses per neuron.
 */
void AllDynamicSTDPSynapses::allocSynapseDeviceStruct( void** allSynapsesDevice, int num_neurons, int maxSynapsesPerNeuron ) {
	AllDynamicSTDPSynapsesDeviceProperties allSynapses;

	allocDeviceStruct( allSynapses, num_neurons, maxSynapsesPerNeuron );

	HANDLE_ERROR( hipMalloc( allSynapsesDevice, sizeof( AllDynamicSTDPSynapsesDeviceProperties ) ) );
	HANDLE_ERROR( hipMemcpy ( *allSynapsesDevice, &allSynapses, sizeof( AllDynamicSTDPSynapsesDeviceProperties ), hipMemcpyHostToDevice ) );
}

/*
 *  Allocate GPU memories to store all synapses' states,
 *  and copy them from host to GPU memory.
 *  (Helper function of allocSynapseDeviceStruct)
 *
 *  @param  allSynapsesDevice     Reference to the AllDynamicSTDPSynapsesDeviceProperties struct 
 *                                on device memory.
 *  @param  num_neurons           Number of neurons.
 *  @param  maxSynapsesPerNeuron  Maximum number of synapses per neuron.
 */
void AllDynamicSTDPSynapses::allocDeviceStruct( AllDynamicSTDPSynapsesDeviceProperties &allSynapses, int num_neurons, int maxSynapsesPerNeuron ) {
        AllSTDPSynapses::allocDeviceStruct( allSynapses, num_neurons, maxSynapsesPerNeuron );

        BGSIZE max_total_synapses = maxSynapsesPerNeuron * num_neurons;

        HANDLE_ERROR( hipMalloc( ( void ** ) &allSynapses.lastSpike, max_total_synapses * sizeof( uint64_t ) ) );
	HANDLE_ERROR( hipMalloc( ( void ** ) &allSynapses.r, max_total_synapses * sizeof( BGFLOAT ) ) );
	HANDLE_ERROR( hipMalloc( ( void ** ) &allSynapses.u, max_total_synapses * sizeof( BGFLOAT ) ) );
	HANDLE_ERROR( hipMalloc( ( void ** ) &allSynapses.D, max_total_synapses * sizeof( BGFLOAT ) ) );
	HANDLE_ERROR( hipMalloc( ( void ** ) &allSynapses.U, max_total_synapses * sizeof( BGFLOAT ) ) );
	HANDLE_ERROR( hipMalloc( ( void ** ) &allSynapses.F, max_total_synapses * sizeof( BGFLOAT ) ) );
}

/*
 *  Delete GPU memories.
 *
 *  @param  allSynapsesDevice  Reference to the AllDynamicSTDPSynapsesDeviceProperties struct 
 *                             on device memory.
 *  @param  sim_info           SimulationInfo to refer from.
 */
void AllDynamicSTDPSynapses::deleteSynapseDeviceStruct( void* allSynapsesDevice ) {
	AllDynamicSTDPSynapsesDeviceProperties allSynapses;

	HANDLE_ERROR( hipMemcpy ( &allSynapses, allSynapsesDevice, sizeof( AllDynamicSTDPSynapsesDeviceProperties ), hipMemcpyDeviceToHost ) );

	deleteDeviceStruct( allSynapses );

	HANDLE_ERROR( hipFree( allSynapsesDevice ) );
}

/*
 *  Delete GPU memories.
 *  (Helper function of deleteSynapseDeviceStruct)
 *
 *  @param  allSynapsesDevice  Reference to the AllDynamicSTDPSynapsesDeviceProperties struct 
 *                             on device memory.
 */
void AllDynamicSTDPSynapses::deleteDeviceStruct( AllDynamicSTDPSynapsesDeviceProperties& allSynapses ) {
        HANDLE_ERROR( hipFree( allSynapses.lastSpike ) );
	HANDLE_ERROR( hipFree( allSynapses.r ) );
	HANDLE_ERROR( hipFree( allSynapses.u ) );
	HANDLE_ERROR( hipFree( allSynapses.D ) );
	HANDLE_ERROR( hipFree( allSynapses.U ) );
	HANDLE_ERROR( hipFree( allSynapses.F ) );

        AllSTDPSynapses::deleteDeviceStruct( allSynapses );
}

/*
 *  Copy all synapses' data from host to device.
 *
 *  @param  allSynapsesDevice  Reference to the AllDynamicSTDPSynapsesDeviceProperties struct 
 *                             on device memory.
 *  @param  sim_info           SimulationInfo to refer from.
 */
void AllDynamicSTDPSynapses::copySynapseHostToDevice( void* allSynapsesDevice, const SimulationInfo *sim_info ) { // copy everything necessary
	copySynapseHostToDevice( allSynapsesDevice, sim_info->totalNeurons, sim_info->maxSynapsesPerNeuron );	
}

/*
 *  Copy all synapses' data from host to device.
 *
 *  @param  allSynapsesDevice     Reference to the AllDynamicSTDPSynapsesDeviceProperties struct 
 *                                on device memory.
 *  @param  num_neurons           Number of neurons.
 *  @param  maxSynapsesPerNeuron  Maximum number of synapses per neuron.
 */
void AllDynamicSTDPSynapses::copySynapseHostToDevice( void* allSynapsesDevice, int num_neurons, int maxSynapsesPerNeuron ) { // copy everything necessary
	AllDynamicSTDPSynapsesDeviceProperties allSynapses;

        HANDLE_ERROR( hipMemcpy ( &allSynapses, allSynapsesDevice, sizeof( AllDynamicSTDPSynapsesDeviceProperties ), hipMemcpyDeviceToHost ) );

	copyHostToDevice( allSynapsesDevice, allSynapses, num_neurons, maxSynapsesPerNeuron );	
}

/*
 *  Copy all synapses' data from host to device.
 *  (Helper function of copySynapseHostToDevice)
 *
 *  @param  allSynapsesDevice     Reference to the AllDynamicSTDPSynapsesDeviceProperties struct 
 *                                on device memory.
 *  @param  num_neurons           Number of neurons.
 *  @param  maxSynapsesPerNeuron  Maximum number of synapses per neuron.
 */
void AllDynamicSTDPSynapses::copyHostToDevice( void* allSynapsesDevice, AllDynamicSTDPSynapsesDeviceProperties& allSynapses, int num_neurons, int maxSynapsesPerNeuron ) { // copy everything necessary 
        AllSTDPSynapses::copyHostToDevice( allSynapsesDevice, allSynapses, num_neurons, maxSynapsesPerNeuron );

        BGSIZE max_total_synapses = maxSynapsesPerNeuron * num_neurons;
        
        HANDLE_ERROR( hipMemcpy ( allSynapses.lastSpike, lastSpike,
                max_total_synapses * sizeof( uint64_t ), hipMemcpyHostToDevice ) );
        HANDLE_ERROR( hipMemcpy ( allSynapses.r, r,
                max_total_synapses * sizeof( BGFLOAT ), hipMemcpyHostToDevice ) );
        HANDLE_ERROR( hipMemcpy ( allSynapses.u, u,
                max_total_synapses * sizeof( BGFLOAT ), hipMemcpyHostToDevice ) );
        HANDLE_ERROR( hipMemcpy ( allSynapses.D, D,
                max_total_synapses * sizeof( BGFLOAT ), hipMemcpyHostToDevice ) );
        HANDLE_ERROR( hipMemcpy ( allSynapses.U, U,
                max_total_synapses * sizeof( BGFLOAT ), hipMemcpyHostToDevice ) );
        HANDLE_ERROR( hipMemcpy ( allSynapses.F, F,
                max_total_synapses * sizeof( BGFLOAT ), hipMemcpyHostToDevice ) );
}

/*
 *  Copy all synapses' data from device to host.
 *
 *  @param  allSynapsesDevice  Reference to the AllDynamicSTDPSynapsesDeviceProperties struct 
 *                             on device memory.
 *  @param  sim_info           SimulationInfo to refer from.
 */
void AllDynamicSTDPSynapses::copySynapseDeviceToHost( void* allSynapsesDevice, const SimulationInfo *sim_info ) {
	// copy everything necessary
	AllDynamicSTDPSynapsesDeviceProperties allSynapses;

        HANDLE_ERROR( hipMemcpy ( &allSynapses, allSynapsesDevice, sizeof( AllDynamicSTDPSynapsesDeviceProperties ), hipMemcpyDeviceToHost ) );

	copyDeviceToHost( allSynapses, sim_info );
}

/*
 *  Copy all synapses' data from device to host.
 *  (Helper function of copySynapseDeviceToHost)
 *
 *  @param  allSynapsesDevice     Reference to the AllDynamicSTDPSynapsesDeviceProperties struct 
 *                                on device memory.
 *  @param  num_neurons           Number of neurons.
 *  @param  maxSynapsesPerNeuron  Maximum number of synapses per neuron.
 */
void AllDynamicSTDPSynapses::copyDeviceToHost( AllDynamicSTDPSynapsesDeviceProperties& allSynapses, const SimulationInfo *sim_info ) {
        AllSTDPSynapses::copyDeviceToHost( allSynapses, sim_info ) ;

	int num_neurons = sim_info->totalNeurons;
	BGSIZE max_total_synapses = sim_info->maxSynapsesPerNeuron * num_neurons;

        HANDLE_ERROR( hipMemcpy ( lastSpike, allSynapses.lastSpike,
                max_total_synapses * sizeof( uint64_t ), hipMemcpyDeviceToHost ) );
        HANDLE_ERROR( hipMemcpy ( r, allSynapses.r,
                max_total_synapses * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
        HANDLE_ERROR( hipMemcpy ( u, allSynapses.u,
                max_total_synapses * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
        HANDLE_ERROR( hipMemcpy ( D, allSynapses.D,
                max_total_synapses * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
        HANDLE_ERROR( hipMemcpy ( U, allSynapses.U,
                max_total_synapses * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
        HANDLE_ERROR( hipMemcpy ( F, allSynapses.F,
                max_total_synapses * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
}

/**     
 *  Set synapse class ID defined by enumClassSynapses for the caller's Synapse class.
 *  The class ID will be set to classSynapses_d in device memory,
 *  and the classSynapses_d will be referred to call a device function for the
 *  particular synapse class.
 *  Because we cannot use virtual function (Polymorphism) in device functions,
 *  we use this scheme.
 *  Note: we used to use a function pointer; however, it caused the growth_cuda crash
 *  (see issue#137).
 */
void AllDynamicSTDPSynapses::setSynapseClassID()
{
    enumClassSynapses classSynapses_h = classAllDynamicSTDPSynapses;

    HANDLE_ERROR( hipMemcpyToSymbol(HIP_SYMBOL(classSynapses_d), &classSynapses_h, sizeof(enumClassSynapses)) );
}

