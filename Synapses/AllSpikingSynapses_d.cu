/*
 * AllSpikingSynapses.cu
 *
 */

#include "AllSpikingSynapses.h"
#include "AllSynapsesDeviceFuncs.h"
#include "Book.h"

/*
 *  Allocate GPU memories to store all synapses' states,
 *  and copy them from host to GPU memory.
 *
 *  @param  allSynapsesDevice  Reference to the AllSpikingSynapsesDeviceProperties struct 
 *                             on device memory.
 *  @param  sim_info           SimulationInfo to refer from.
 */
void AllSpikingSynapses::allocSynapseDeviceStruct( void** allSynapsesDevice, const SimulationInfo *sim_info ) {
        allocSynapseDeviceStruct( allSynapsesDevice, sim_info->totalNeurons, sim_info->maxSynapsesPerNeuron );
}

/*
 *  Allocate GPU memories to store all synapses' states,
 *  and copy them from host to GPU memory.
 *
 *  @param  allSynapsesDevice     Reference to the AllSpikingSynapsesDeviceProperties struct 
 *                                on device memory.
 *  @param  num_neurons           Number of neurons.
 *  @param  maxSynapsesPerNeuron  Maximum number of synapses per neuron.
 */
void AllSpikingSynapses::allocSynapseDeviceStruct( void** allSynapsesDevice, int num_neurons, int maxSynapsesPerNeuron ) {
        AllSpikingSynapsesDeviceProperties allSynapses;

        allocDeviceStruct( allSynapses, num_neurons, maxSynapsesPerNeuron );

        HANDLE_ERROR( hipMalloc( allSynapsesDevice, sizeof( AllSpikingSynapsesDeviceProperties ) ) );
        HANDLE_ERROR( hipMemcpy ( *allSynapsesDevice, &allSynapses, sizeof( AllSpikingSynapsesDeviceProperties ), hipMemcpyHostToDevice ) );
}

/*
 *  Allocate GPU memories to store all synapses' states,
 *  and copy them from host to GPU memory.
 *  (Helper function of allocSynapseDeviceStruct)
 *
 *  @param  allSynapsesDevice     Reference to the AllSpikingSynapsesDeviceProperties struct 
 *                                on device memory.
 *  @param  num_neurons           Number of neurons.
 *  @param  maxSynapsesPerNeuron  Maximum number of synapses per neuron.
 */
void AllSpikingSynapses::allocDeviceStruct( AllSpikingSynapsesDeviceProperties &allSynapses, int num_neurons, int maxSynapsesPerNeuron ) {
        BGSIZE max_total_synapses = maxSynapsesPerNeuron * num_neurons;

        HANDLE_ERROR( hipMalloc( ( void ** ) &allSynapses.sourceNeuronIndex, max_total_synapses * sizeof( int ) ) );
        HANDLE_ERROR( hipMalloc( ( void ** ) &allSynapses.destNeuronIndex, max_total_synapses * sizeof( int ) ) );
        HANDLE_ERROR( hipMalloc( ( void ** ) &allSynapses.W, max_total_synapses * sizeof( BGFLOAT ) ) );
        HANDLE_ERROR( hipMalloc( ( void ** ) &allSynapses.type, max_total_synapses * sizeof( synapseType ) ) );
        HANDLE_ERROR( hipMalloc( ( void ** ) &allSynapses.psr, max_total_synapses * sizeof( BGFLOAT ) ) );
        HANDLE_ERROR( hipMalloc( ( void ** ) &allSynapses.in_use, max_total_synapses * sizeof( bool ) ) );
        HANDLE_ERROR( hipMalloc( ( void ** ) &allSynapses.synapse_counts, num_neurons * sizeof( BGSIZE ) ) );
        HANDLE_ERROR( hipMalloc( ( void ** ) &allSynapses.decay, max_total_synapses * sizeof( BGFLOAT ) ) );
        HANDLE_ERROR( hipMalloc( ( void ** ) &allSynapses.tau, max_total_synapses * sizeof( BGFLOAT ) ) );
        HANDLE_ERROR( hipMalloc( ( void ** ) &allSynapses.total_delay, max_total_synapses * sizeof( int ) ) );
        HANDLE_ERROR( hipMalloc( ( void ** ) &allSynapses.delayQueue, max_total_synapses * sizeof( uint32_t ) ) );
        HANDLE_ERROR( hipMalloc( ( void ** ) &allSynapses.delayIdx, max_total_synapses * sizeof( int ) ) );
        HANDLE_ERROR( hipMalloc( ( void ** ) &allSynapses.ldelayQueue, max_total_synapses * sizeof( int ) ) );
}

/*
 *  Delete GPU memories.
 *
 *  @param  allSynapsesDevice  Reference to the AllSpikingSynapsesDeviceProperties struct 
 *                             on device memory.
 *  @param  sim_info           SimulationInfo to refer from.
 */
void AllSpikingSynapses::deleteSynapseDeviceStruct( void* allSynapsesDevice ) {
        AllSpikingSynapsesDeviceProperties allSynapses;

        HANDLE_ERROR( hipMemcpy ( &allSynapses, allSynapsesDevice, sizeof( AllSpikingSynapsesDeviceProperties ), hipMemcpyDeviceToHost ) );

        deleteDeviceStruct( allSynapses );

        HANDLE_ERROR( hipFree( allSynapsesDevice ) );
}

/*
 *  Delete GPU memories.
 *  (Helper function of deleteSynapseDeviceStruct)
 *
 *  @param  allSynapsesDevice  Reference to the AllSpikingSynapsesDeviceProperties struct 
 *                             on device memory.
 */
void AllSpikingSynapses::deleteDeviceStruct( AllSpikingSynapsesDeviceProperties& allSynapses ) {
        HANDLE_ERROR( hipFree( allSynapses.sourceNeuronIndex ) );
        HANDLE_ERROR( hipFree( allSynapses.destNeuronIndex ) );
        HANDLE_ERROR( hipFree( allSynapses.W ) );
        HANDLE_ERROR( hipFree( allSynapses.type ) );
        HANDLE_ERROR( hipFree( allSynapses.psr ) );
        HANDLE_ERROR( hipFree( allSynapses.in_use ) );
        HANDLE_ERROR( hipFree( allSynapses.synapse_counts ) );
        HANDLE_ERROR( hipFree( allSynapses.decay ) );
        HANDLE_ERROR( hipFree( allSynapses.tau ) );
        HANDLE_ERROR( hipFree( allSynapses.total_delay ) );
        HANDLE_ERROR( hipFree( allSynapses.delayQueue ) );
        HANDLE_ERROR( hipFree( allSynapses.delayIdx ) );
        HANDLE_ERROR( hipFree( allSynapses.ldelayQueue ) );

        // Set count_neurons to 0 to avoid illegal memory deallocation 
        // at AllSpikingSynapses deconstructor.
        //allSynapses.count_neurons = 0;
}

/*
 *  Copy all synapses' data from host to device.
 *
 *  @param  allSynapsesDevice  Reference to the AllSpikingSynapsesDeviceProperties struct 
 *                             on device memory.
 *  @param  sim_info           SimulationInfo to refer from.
 */
void AllSpikingSynapses::copySynapseHostToDevice( void* allSynapsesDevice, const SimulationInfo *sim_info ) { // copy everything necessary
        copySynapseHostToDevice( allSynapsesDevice, sim_info->totalNeurons, sim_info->maxSynapsesPerNeuron );
}

/*
 *  Copy all synapses' data from host to device.
 *
 *  @param  allSynapsesDevice     Reference to the AllSpikingSynapsesDeviceProperties struct 
 *                                on device memory.
 *  @param  num_neurons           Number of neurons.
 *  @param  maxSynapsesPerNeuron  Maximum number of synapses per neuron.
 */
void AllSpikingSynapses::copySynapseHostToDevice( void* allSynapsesDevice, int num_neurons, int maxSynapsesPerNeuron ) { // copy everything necessary
        AllSpikingSynapsesDeviceProperties allSynapses;

        HANDLE_ERROR( hipMemcpy ( &allSynapses, allSynapsesDevice, sizeof( AllSpikingSynapsesDeviceProperties ), hipMemcpyDeviceToHost ) );

        copyHostToDevice( allSynapsesDevice, allSynapses, num_neurons, maxSynapsesPerNeuron );
}

/*
 *  Copy all synapses' data from host to device.
 *  (Helper function of copySynapseHostToDevice)
 *
 *  @param  allSynapsesDevice     Reference to the AllSpikingSynapsesDeviceProperties struct 
 *                                on device memory.
 *  @param  num_neurons           Number of neurons.
 *  @param  maxSynapsesPerNeuron  Maximum number of synapses per neuron.
 */
void AllSpikingSynapses::copyHostToDevice( void* allSynapsesDevice, AllSpikingSynapsesDeviceProperties& allSynapses, int num_neurons, int maxSynapsesPerNeuron ) { // copy everything necessary 
        BGSIZE max_total_synapses = maxSynapsesPerNeuron * num_neurons;

        allSynapses.maxSynapsesPerNeuron = maxSynapsesPerNeuron;
        allSynapses.total_synapse_counts = total_synapse_counts;
        allSynapses.count_neurons = count_neurons;
        HANDLE_ERROR( hipMemcpy ( allSynapsesDevice, &allSynapses, sizeof( AllSpikingSynapsesDeviceProperties ), hipMemcpyHostToDevice ) );

        // Set count_neurons to 0 to avoid illegal memory deallocation 
        // at AllSpikingSynapses deconstructor.
        allSynapses.count_neurons = 0;

        HANDLE_ERROR( hipMemcpy ( allSynapses.sourceNeuronIndex, sourceNeuronIndex,
                max_total_synapses * sizeof( int ), hipMemcpyHostToDevice ) );
        HANDLE_ERROR( hipMemcpy ( allSynapses.destNeuronIndex, destNeuronIndex,
                max_total_synapses * sizeof( int ),  hipMemcpyHostToDevice ) );
        HANDLE_ERROR( hipMemcpy ( allSynapses.W, W,
                max_total_synapses * sizeof( BGFLOAT ), hipMemcpyHostToDevice ) );
        HANDLE_ERROR( hipMemcpy ( allSynapses.type, type,
                max_total_synapses * sizeof( synapseType ), hipMemcpyHostToDevice ) );
        HANDLE_ERROR( hipMemcpy ( allSynapses.psr, psr,
                max_total_synapses * sizeof( BGFLOAT ), hipMemcpyHostToDevice ) );
        HANDLE_ERROR( hipMemcpy ( allSynapses.in_use, in_use,
                max_total_synapses * sizeof( bool ), hipMemcpyHostToDevice ) );
        HANDLE_ERROR( hipMemcpy ( allSynapses.synapse_counts, synapse_counts,
                        num_neurons * sizeof( BGSIZE ), hipMemcpyHostToDevice ) );
        HANDLE_ERROR( hipMemcpy ( allSynapses.decay, decay,
                max_total_synapses * sizeof( BGFLOAT ), hipMemcpyHostToDevice ) );
        HANDLE_ERROR( hipMemcpy ( allSynapses.tau, tau,
                max_total_synapses * sizeof( BGFLOAT ), hipMemcpyHostToDevice ) );
        HANDLE_ERROR( hipMemcpy ( allSynapses.total_delay, total_delay,
                max_total_synapses * sizeof( int ), hipMemcpyHostToDevice ) );
        HANDLE_ERROR( hipMemcpy ( allSynapses.delayQueue, delayQueue,
                max_total_synapses * sizeof( uint32_t ), hipMemcpyHostToDevice ) );
        HANDLE_ERROR( hipMemcpy ( allSynapses.delayIdx, delayIdx,
                max_total_synapses * sizeof( int ), hipMemcpyHostToDevice ) );
        HANDLE_ERROR( hipMemcpy ( allSynapses.ldelayQueue, ldelayQueue,
                max_total_synapses * sizeof( int ), hipMemcpyHostToDevice ) );
}

/*
 *  Copy all synapses' data from device to host.
 *
 *  @param  allSynapsesDevice  Reference to the AllSpikingSynapsesDeviceProperties struct 
 *                             on device memory.
 *  @param  sim_info           SimulationInfo to refer from.
 */
void AllSpikingSynapses::copySynapseDeviceToHost( void* allSynapsesDevice, const SimulationInfo *sim_info ) {
        // copy everything necessary
        AllSpikingSynapsesDeviceProperties allSynapses;

        HANDLE_ERROR( hipMemcpy ( &allSynapses, allSynapsesDevice, sizeof( AllSpikingSynapsesDeviceProperties ), hipMemcpyDeviceToHost ) );

        copyDeviceToHost( allSynapses, sim_info );
}

/*
 *  Copy all synapses' data from device to host.
 *  (Helper function of copySynapseDeviceToHost)
 *
 *  @param  allSynapsesDevice     Reference to the AllSpikingSynapsesDeviceProperties struct 
 *                                on device memory.
 *  @param  num_neurons           Number of neurons.
 *  @param  maxSynapsesPerNeuron  Maximum number of synapses per neuron.
 */
void AllSpikingSynapses::copyDeviceToHost( AllSpikingSynapsesDeviceProperties& allSynapses, const SimulationInfo *sim_info ) {
        int num_neurons = sim_info->totalNeurons;
        BGSIZE max_total_synapses = sim_info->maxSynapsesPerNeuron * num_neurons;

        HANDLE_ERROR( hipMemcpy ( synapse_counts, allSynapses.synapse_counts,
                num_neurons * sizeof( BGSIZE ), hipMemcpyDeviceToHost ) );
        maxSynapsesPerNeuron = allSynapses.maxSynapsesPerNeuron;
        total_synapse_counts = allSynapses.total_synapse_counts;
        count_neurons = allSynapses.count_neurons;

        // Set count_neurons to 0 to avoid illegal memory deallocation 
        // at AllSpikingSynapses deconstructor.
        allSynapses.count_neurons = 0;

        HANDLE_ERROR( hipMemcpy ( sourceNeuronIndex, allSynapses.sourceNeuronIndex,
                max_total_synapses * sizeof( int ), hipMemcpyDeviceToHost ) );
        HANDLE_ERROR( hipMemcpy ( destNeuronIndex, allSynapses.destNeuronIndex,
                max_total_synapses * sizeof( int ), hipMemcpyDeviceToHost ) );
        HANDLE_ERROR( hipMemcpy ( W, allSynapses.W,
                max_total_synapses * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
        HANDLE_ERROR( hipMemcpy ( type, allSynapses.type,
                max_total_synapses * sizeof( synapseType ), hipMemcpyDeviceToHost ) );
        HANDLE_ERROR( hipMemcpy ( psr, allSynapses.psr,
                max_total_synapses * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
        HANDLE_ERROR( hipMemcpy ( in_use, allSynapses.in_use,
                max_total_synapses * sizeof( bool ), hipMemcpyDeviceToHost ) );
        HANDLE_ERROR( hipMemcpy ( decay, allSynapses.decay,
                max_total_synapses * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
        HANDLE_ERROR( hipMemcpy ( tau, allSynapses.tau,
                max_total_synapses * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
        HANDLE_ERROR( hipMemcpy ( total_delay, allSynapses.total_delay,
                max_total_synapses * sizeof( int ), hipMemcpyDeviceToHost ) );
        HANDLE_ERROR( hipMemcpy ( delayQueue, allSynapses.delayQueue,
                max_total_synapses * sizeof( uint32_t ), hipMemcpyDeviceToHost ) );
        HANDLE_ERROR( hipMemcpy ( delayIdx, allSynapses.delayIdx,
                max_total_synapses * sizeof( int ), hipMemcpyDeviceToHost ) );
        HANDLE_ERROR( hipMemcpy ( ldelayQueue, allSynapses.ldelayQueue,
                max_total_synapses * sizeof( int ), hipMemcpyDeviceToHost ) );
}

/*
 *  Get synapse_counts in AllSynapses struct on device memory.
 *
 *  @param  allSynapsesDevice  Reference to the AllSpikingSynapsesDeviceProperties struct 
 *                             on device memory.
 *  @param  sim_info           SimulationInfo to refer from.
 */
void AllSpikingSynapses::copyDeviceSynapseCountsToHost(void* allSynapsesDevice, const SimulationInfo *sim_info)
{
        AllSpikingSynapsesDeviceProperties allSynapses;
        int neuron_count = sim_info->totalNeurons;

        HANDLE_ERROR( hipMemcpy ( &allSynapses, allSynapsesDevice, sizeof( AllSpikingSynapsesDeviceProperties ), hipMemcpyDeviceToHost ) );
        HANDLE_ERROR( hipMemcpy ( synapse_counts, allSynapses.synapse_counts, neuron_count * sizeof( BGSIZE ), hipMemcpyDeviceToHost ) );

        // Set count_neurons to 0 to avoid illegal memory deallocation 
        // at AllSpikingSynapses deconstructor.
        //allSynapses.count_neurons = 0;
}

/* 
 *  Get summationCoord and in_use in AllSynapses struct on device memory.
 *
 *  @param  allSynapsesDevice  Reference to the AllSpikingSynapsesDeviceProperties struct 
 *                             on device memory.
 *  @param  sim_info           SimulationInfo to refer from.
 */
void AllSpikingSynapses::copyDeviceSynapseSumIdxToHost(void* allSynapsesDevice, const SimulationInfo *sim_info)
{
        AllSpikingSynapsesDeviceProperties allSynapses;
        BGSIZE max_total_synapses = sim_info->maxSynapsesPerNeuron * sim_info->totalNeurons;

        HANDLE_ERROR( hipMemcpy ( &allSynapses, allSynapsesDevice, sizeof( AllSpikingSynapsesDeviceProperties ), hipMemcpyDeviceToHost ) );
        HANDLE_ERROR( hipMemcpy ( sourceNeuronIndex, allSynapses.sourceNeuronIndex,
                max_total_synapses * sizeof( int ), hipMemcpyDeviceToHost ) );
        HANDLE_ERROR( hipMemcpy ( in_use, allSynapses.in_use,
                max_total_synapses * sizeof( bool ), hipMemcpyDeviceToHost ) );
       
        // Set count_neurons to 0 to avoid illegal memory deallocation 
        // at AllSpikingSynapses deconstructor.
        //allSynapses.count_neurons = 0;
}

/*
 *  Set some parameters used for advanceSynapsesDevice.
 */
void AllSpikingSynapses::setAdvanceSynapsesDeviceParams()
{
    setSynapseClassID();
}

/**
 *  Set synapse class ID defined by enumClassSynapses for the caller's Synapse class.
 *  The class ID will be set to classSynapses_d in device memory,
 *  and the classSynapses_d will be referred to call a device function for the
 *  particular synapse class.
 *  Because we cannot use virtual function (Polymorphism) in device functions,
 *  we use this scheme.
 *  Note: we used to use a function pointer; however, it caused the growth_cuda crash
 *  (see issue#137).
 */
void AllSpikingSynapses::setSynapseClassID()
{
    enumClassSynapses classSynapses_h = classAllSpikingSynapses;

    HANDLE_ERROR( hipMemcpyToSymbol(HIP_SYMBOL(classSynapses_d), &classSynapses_h, sizeof(enumClassSynapses)) );
}

/*
 *  Advance all the Synapses in the simulation.
 *  Update the state of all synapses for a time step.
 *
 *  @param  allSynapsesDevice      Reference to the AllSynapsesDeviceProperties struct 
 *                                 on device memory.
 *  @param  allNeuronsDevice       Reference to the allNeurons struct on device memory.
 *  @param  synapseIndexMapDevice  Reference to the SynapseIndexMap on device memory.
 *  @param  sim_info               SimulationInfo class to read information from.
 */
void AllSpikingSynapses::advanceSynapses(void* allSynapsesDevice, void* allNeuronsDevice, void* synapseIndexMapDevice, const SimulationInfo *sim_info)
{
    if (total_synapse_counts == 0)
        return;

    // CUDA parameters
    const int threadsPerBlock = 256;
    int blocksPerGrid = ( total_synapse_counts + threadsPerBlock - 1 ) / threadsPerBlock;

    // Advance synapses ------------->
    advanceSpikingSynapsesDevice <<< blocksPerGrid, threadsPerBlock >>> ( total_synapse_counts, (SynapseIndexMap*)synapseIndexMapDevice, g_simulationStep, sim_info->deltaT, (AllSpikingSynapsesDeviceProperties*)allSynapsesDevice );
}
